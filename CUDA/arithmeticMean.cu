#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define N 512

__global__ void ArithmeticMean (int *a, int *o)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    for(int i = N/2; i > 0; i = i/2)
    {
        if(tid < i)
        {
            a[tid]+=a[tid+i];
        }
    }
    o[0] = a[0];
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a;

    int size = N*sizeof(int);

    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);

    for(int i = 1; i <= N ; i++)
    {
        h_a[i-1] = i;
    }

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    ArithmeticMean<<<1, N/2>>>(d_a,o_a);

    hipDeviceSynchronize();

    hipMemcpy(h_a,d_a,size,hipMemcpyDeviceToHost);
    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);
    
    float AM =(float) oh_a[0]/N;
    printf("Arithmatic Mean is %.2f\n", AM);

    hipFree(d_a);
    free(h_a);

    return 0;
}