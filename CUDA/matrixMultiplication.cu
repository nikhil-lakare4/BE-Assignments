#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define N 10

__global__ void fMatrixMultiplication (int *A,int *B,int *C)
{
  int ROW = blockIdx.y*blockDim.y+threadIdx.y;
  int COL = blockIdx.x*blockDim.x+threadIdx.x;

  int tmpSum = 0;

  if (ROW < N && COL < N) {
      // each thread computes one element of the block sub-matrix
      for (int i = 0; i < N; i++) {
          tmpSum += A[ROW * N + i] * B[i * N + COL];
      }
      C[ROW * N + COL] = tmpSum;
    }
}

int main()
{
    int *h_A,*h_B,*h_C;

    int cnt=1;

    int size = N*N*sizeof(int);

    h_A = (int*) malloc(size);
    h_B = (int*) malloc(size);
    h_C = (int*) malloc(size);

    // Initialize matrices on the host
    for (int i=1; i<=N*N; i++){
      if(cnt==N+1)
        cnt = 1;
      h_A[i-1] = cnt;
      h_B[i-1] = cnt;
      cnt+=1;
    }

    printf("Matrix A\n");
    for (int i=0; i<N*N; i++){
	     printf("%d", h_A[i]);
	     if(i%N==N-1)
		    printf("\n");
    }
    printf("\n");

    printf("Matrix B\n");
    for (int i=0; i<N*N; i++){
	     printf("%d", h_B[i]);
	     if(i%N==N-1)
		      printf("\n");
    }
    printf("\n");

    int *d_A,*d_B,*d_C;

    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);

    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
     
    fMatrixMultiplication<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,d_C);

    hipDeviceSynchronize();

    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
    
    printf("Matrix C = A*B\n");
        
    for (int i=0; i<N*N; i++){
        cout<<h_C[i]<<" ";
        if(i%N==N-1)
          printf("\n");
    }

    hipFree(d_A);
    free(h_A);
    hipFree(d_B);
    free(h_B);
    hipFree(d_C);
    free(h_C);

    return 0;
}