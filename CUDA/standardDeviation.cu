#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define N 512

__global__ void Sum (int *a,int *o)
{

    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    for(int i = N/2; i > 0; i = i/2)
    {
        if(tid < i)
        {
            a[tid]+=a[tid+i];
        }

    }

    o[0] = a[0];
}

__global__ void standardDeviation(int *a,int avg)
{
  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  if(tid<N)
  {
    a[tid] -= avg;
    a[tid] = a[tid]*a[tid];
  }
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a,*d_a1;

    int size = N*sizeof(int);

    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);
    //new
    hipMalloc((void**)&d_a1,size);

    for(int i = 1; i <= N; i++)
    {
        h_a[i-1] = i;
    }


    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_a1,h_a,size,hipMemcpyHostToDevice);

    Sum<<<1,N/2>>>(d_a,o_a);

    hipDeviceSynchronize();

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    int arithmetcMean = oh_a[0]/N;

    standardDeviation<<<1,N>>>(d_a1,arithmetcMean);

    Sum<<<1,N/2>>>(d_a1,o_a);

    hipDeviceSynchronize();

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    int tmp = oh_a[0]/N;

    printf("Standard Deviation is - %.2f\n", sqrt(tmp));

    hipFree(d_a);
    free(h_a);
    hipFree(o_a);
    free(oh_a);
    hipFree(d_a1);

    return 0;
}

