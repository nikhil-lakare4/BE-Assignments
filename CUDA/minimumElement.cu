#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define N 512

__global__ void minElement (int *a, int *o)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    for(int i = N/2; i > 0; i = i/2)
    {
        if(tid < i)
        {
            if(a[tid] > a[tid+i])
            {
                a[tid] = a[tid+i];
            }
        }
    }

    o[0] = a[0];
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a;

    int size = N*sizeof(int);

    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);

    for(int i = N; i >= 1; i--)
    {
        h_a[i-1] = i;
    }

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    minElement<<<1,N/2>>>(d_a,o_a);

    hipDeviceSynchronize();

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    printf("Min Element is - %d\n", oh_a[0]);

    hipFree(d_a);
    free(h_a);

    return 0;
}

