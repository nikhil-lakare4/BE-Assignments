#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 512

__global__ void bmk_add(int* a, int* b, int* result)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	result[i] = a[i] + b[i];
}

int main()
{
	int* a, * b, * c;
	int* dev_a, * dev_b, * dev_c;

	int size = N * sizeof(int);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i;
	}

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	bmk_add << <1, 512 >> > (dev_a, dev_b, dev_c);

	hipDeviceSynchronize();

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
		printf("%d  ", c[i]);

	printf("\n");
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	free(a);
	free(b);
	free(c);

	return 0;
}