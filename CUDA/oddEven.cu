#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
static const int WORK_SIZE = 10;

__global__ void sort(int* a, int i, int n)
{
	int tid = threadIdx.x;
	int p;
	int temp;
	if (i % 2 == 0)
	{
		p = tid * 2;

		if (a[p] > a[p + 1])
		{
			temp = a[p];
			a[p] = a[p + 1];
			a[p + 1] = temp;
		}
	}
	else
	{
		p = tid * 2 + 1;

		if (p < n - 1)
		{
			if (a[p] > a[p + 1])
			{
				temp = a[p];
				a[p] = a[p + 1];
				a[p + 1] = temp;
			}
		}
	}
}

int main(void)
{
	int a[WORK_SIZE];
	int i;
	int* da;

	hipMalloc((void**)&da, sizeof(int) * WORK_SIZE);

	for (i = 0; i < WORK_SIZE; i++)
	{
		printf("%d:", i);
		scanf("%d", &a[i]);
	}

	hipMemcpy(da, a, sizeof(int) * WORK_SIZE,
		hipMemcpyHostToDevice);

	for (i = 0; i < WORK_SIZE; i++)
	{
		sort << <1, WORK_SIZE / 2 >> > (da, i, WORK_SIZE);
	}
	hipDeviceSynchronize(); // Wait for the GPU launched work to complete
	hipGetLastError();

	hipMemcpy(a, da, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost);

	for (i = 0; i < WORK_SIZE; i++)
	{
		printf("%d\t", a[i]);
	}
	printf("\n");
	hipFree((void*)da);
	return 0;
}